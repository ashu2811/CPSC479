#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


//this is the function that finds the min within the matrix
__global__ void getminimum(unsigned *da, unsigned* minValue){
  int i = threadIdx.x * blockDim.y + threadIdx.y;
  atomicMin(minValue, da[i]);
}



//fill matrix with random nums
__global__ void generateRandomMatrix(unsigned *da, float x, int n)
{
  hiprandState_t state;
  hiprand_init(clock64(), 0, 0, &state);
  int i = threadIdx.x * blockDim.y + threadIdx.y;
  da[i] = hiprand(&state) % 100 + i;
}


//set min as max int value
__global__ void makeMax(unsigned *min){
   min[0] = INT_MAX; 
}


__global__ void square ( unsigned *matrix, unsigned *result, unsigned matrixsize) {
  unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
  for (unsigned jj = 0; jj < matrixsize; jj++) {
    for (unsigned kk = 0; kk < matrixsize; kk++) {
      result[id * matrixsize + jj] += matrix[id * matrixsize + kk] * matrix[kk * matrixsize + jj]; 
     }
   }
}


__global__ void initializeResult(unsigned *result, unsigned size){
  for(unsigned i = 0; i < size; i++){
    result[i] = 0;
 }
}



//PART 1 findMIN
__host__ void findMin(unsigned *matrix, dim3 block){
  unsigned *min, *C_min;                              //store the min number
  min = (unsigned *)malloc(1 * sizeof(unsigned));     //allocate space for min
  hipMalloc((void **)&C_min, 1 * sizeof(unsigned));  //allocate space for device copy
  hipMemcpy(C_min, min, 1 * sizeof(unsigned), hipMemcpyHostToDevice);  //copy input to device 
  makeMax<<<1,1>>>(C_min);                            //set min value as a MAX for initial
  getminimum<<<1, block>>>(matrix, C_min);             //find the min within matrix
  hipMemcpy(min, C_min, 1 * sizeof(unsigned), hipMemcpyDeviceToHost); //copy back to host
  printf("Min Value is %d\n", min[0]);                  //print min
  free(min);
  hipFree(C_min);
}




#define N 16
//PART 2 findSquare
__host__ void findSquare(unsigned *matrix){
  unsigned *result, *hresult;                                  //store the square
  hresult = (unsigned *)malloc(N * N  * sizeof(unsigned));     //allocate space for square
  hipMalloc(&result, N * N  * sizeof(unsigned));              //allocate space for device copy
  initializeResult<<<1, N>>>(result, N);                       
  square<<<1, N>>>(matrix, result, N);                         //set square for n 
  hipMemcpy(hresult, result, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);  //copy back to host
  printf("Resulting square Matrix\n");             //print square
  for(unsigned ii = 0; ii < N; ++ii){
    for(unsigned jj = 0; jj < N; ++jj){
      //printf("%2d ", hresult[ii * N + jj]);
    }
    printf("\n");
 }
}


int main() {
    dim3 block(N, N, 1);
    unsigned *matrix, *hmatrix;                         //hold the matrix
    hipMalloc(&matrix, N * N * sizeof(unsigned));      //alloc matrix
    hmatrix = (unsigned *)malloc(N * N * sizeof(unsigned));  //alloc matrix
    generateRandomMatrix<<<1, block>>>(matrix, 5, 30);  //fill matrix with random numbers
    hipMemcpy(hmatrix, matrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);
    printf("starting matrix\n");
      for (unsigned ii = 0; ii < N; ++ii) {
        for (unsigned jj = 0; jj < N; ++jj) {
           // printf("%2d ", hmatrix[ii * N + jj]);
        }
        printf("\n");
   }
   findMin(matrix, block);  
   findSquare(matrix);
   return 0;
}
